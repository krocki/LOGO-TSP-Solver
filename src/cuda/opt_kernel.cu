#include "hip/hip_runtime.h"
/*
 *   Logo TSP Solver ver. 0.62  Copyright (C) 2013  Kamil Rocki
 *
 *   This program is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   This program is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <headers.h>
#include <cuda_common.h>
#include <cuda_defs.h>

#ifdef LEGACY_GPUS
#define REDUCE_ON_CPU
#endif

#ifndef USE_ORDERED_COORDS
__launch_bounds__ (1024, 32)
__global__ void kernel2opt (city_coords * device_coords, int _cities,
                            ROUTE_DATA_TYPE * device_route,
                            unsigned long number_of_2opts,
                            unsigned int iterations, int *mutex,
                            struct best2_out *best_2opt) {
#else
__launch_bounds__ (1024, 32)
__global__ void kernel2opt (city_coords * device_coords, int _cities,
                            unsigned long number_of_2opts,
                            unsigned int iterations, int *mutex,
                            struct best2_out *best_2opt) {
#endif
    register int    local_id = threadIdx.x + blockIdx.x * blockDim.x;
    register int    id;
    register unsigned int i,
             j;
    register unsigned long max = number_of_2opts;
    // 2-opt move index
    register int    change;
    register int    packSize = blockDim.x * gridDim.x;
    struct best2_out best;
    register int    iter = iterations;
    best.minchange = 999999;
#ifdef LEGACY_GPUS
    __shared__ city_coords coords[CUDA_MAX_SIZE_SIMPLE_LEGACY];
#else
    __shared__ city_coords coords[CUDA_MAX_SIZE_SIMPLE];
#endif
    __shared__ int  cities;
#ifdef LEGACY_GPUS
    __shared__ best2_out best_values[256];
#else
    __shared__ best2_out best_values[1024];
#endif
    // copy to a local register
    cities = _cities;
    // copy the coordinates of all route points to the shared memory
#ifndef USE_ORDERED_COORDS

    for (register int k = threadIdx.x; k < cities; k += blockDim.x) {
        coords[k] = device_coords[device_route[k]];
    }

#else

    for (register int k = threadIdx.x; k < cities; k += blockDim.x) {
        coords[k] = device_coords[k];
    }

#endif
    __syncthreads();
    // each thread performs iter inner iterations in order to reuse the shared
    // memory
#pragma unroll

    for (register int no = 0; no < iter; no++) {
        id = local_id + no * packSize;

        if (id < max) {
            // a nasty formula to calculate the right cell of a triangular matrix
            // index based on the thread id
            j = (unsigned int) (3 + __fsqrt_rn (8.0f * (float) id + 1.0f) ) / 2;	// l_idx.i;
            i = id - (j - 2) * (j - 1) / 2 + 1;
            // calculate the effect of (i,j) swap
            change =
                calculateDistance2DSimple (i, j + 1,
                                           coords) + calculateDistance2DSimple (i - 1,
                                                   j,
                                                   coords)
                - calculateDistance2DSimple (i, i - 1,
                                             coords) - calculateDistance2DSimple (j +
                                                     1, j,
                                                     coords);

            // save if best than already known swap
            if (change < best.minchange) {
                best.minchange = change;
                best.i = i;
                best.j = j + 1;
            }
        }
    }

    // intra-block reduction...it doesn't look pretty, but works
    best_values[threadIdx.x] = best;
    __syncthreads();
#ifndef LEGACY_GPUS

    // 1024 threads
    if (threadIdx.x < 512)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 512].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 512].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 512].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 512].j;
        }

    __syncthreads();

    // 512 threads
    if (threadIdx.x < 256)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 256].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 256].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 256].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 256].j;
        }

    __syncthreads();
#endif

    // 256 threads
    if (threadIdx.x < 128)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 128].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 128].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 128].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 128].j;
        }

    __syncthreads();

    // 128 threads
    if (threadIdx.x < 64)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 64].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 64].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 64].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 64].j;
        }

    __syncthreads();

    // 64 threads
    if (threadIdx.x < 32)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 32].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 32].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 32].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 32].j;
        }

    __syncthreads();

    // 32 threads
    if (threadIdx.x < 16)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 16].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 16].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 16].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 16].j;
        }

    __syncthreads();

    // 16 threads
    if (threadIdx.x < 8)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 8].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 8].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 8].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 8].j;
        }

    __syncthreads();

    // 8 threads
    if (threadIdx.x < 4)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 4].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 4].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 4].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 4].j;
        }

    __syncthreads();

    // 4 threads
    if (threadIdx.x < 2)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 2].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 2].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 2].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 2].j;
        }

    __syncthreads();

    // 2 threads
    if (threadIdx.x < 1)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 1].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 1].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 1].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 1].j;
        }

    __syncthreads();

    // 1 thread left
    // 1st thread in a block holds the best result
    if (threadIdx.x == 0) {
#ifndef REDUCE_ON_CPU
        // reduction on GPU

        if (best_values[threadIdx.x].minchange < best_2opt->minchange) {
            // inter-block reduction
            atomicMin (& (best_2opt->minchange), best_values[threadIdx.x].minchange);
            // synchronize
            lock (mutex);

            if (best_values[threadIdx.x].minchange == best_2opt->minchange) {
                best_2opt->i = best_values[threadIdx.x].i;
                best_2opt->j = best_values[threadIdx.x].j;
            }

            unlock (mutex);
        }

#else
        // reduction on host
        best_2opt[blockIdx.x] = best_values[threadIdx.x];
#endif
    }
}

// currently this code is hard-coded for 16x1024 thread configuration
// the subproblem block size is 2048x2048 with 128x128 thread groups iterating 16x16
// times (16 times each direction)
#ifndef USE_ORDERED_COORDS
__launch_bounds__ (1024, 64)
__global__ void kernel2opt_extended (city_coords * coords_global,
                                     ROUTE_DATA_TYPE * device_route, int _cities,
                                     int *mutex, struct best2_out *best_2opt) {
#else
__launch_bounds__ (1024, 64)
__global__ void kernel2opt_extended (city_coords * coords_global, int _cities,
                                     int *mutex, struct best2_out *best_2opt) {
#endif
    register int    local_id = threadIdx.x + blockIdx.x * blockDim.x;
    register int    i,
             j;
    register int    change;
    struct best2_out best;
    register int    rA_start,
             rB_start,
             rA_end,
             rB_end;
    // 2 ranges of coordinates
#ifndef LEGACY_GPUS
    __shared__ city_coords coords_extended_A[CUDA_MAX_COORDS_EXTENDED + 1];
    __shared__ city_coords coords_extended_B[CUDA_MAX_COORDS_EXTENDED + 1];
#else
    __shared__ city_coords coords_extended_A[CUDA_MAX_COORDS_EXTENDED_LEGACY + 1];
    __shared__ city_coords coords_extended_B[CUDA_MAX_COORDS_EXTENDED_LEGACY + 1];
#endif
    __shared__ int  cities;
#ifdef LEGACY_GPUS
    __shared__ best2_out best_values[256];
#else
    __shared__ best2_out best_values[1024];
#endif
    cities = _cities;
#ifdef LEGACY_GPUS
#define OFFSET 512
#else
#define OFFSET 2048
#endif
    register int    const1 = cities - OFFSET;
#ifdef K20
    register int    const2 = (cities - (cities / OFFSET) * OFFSET) / 256 + 1;
#else
    register int    const2 = (cities - (cities / OFFSET) * OFFSET) / 128 + 1;
#endif
#ifdef LEGACY_GPUS
    register int    iters_inside_a_blockB = 4;
#else
#ifndef K20
    register int    iters_inside_a_blockB = 16;
#else
    register int    iters_inside_a_blockB = 8;
#endif
#endif
    best.minchange = 99999;
    best.i = 1;
    best.j = 1;
    __syncthreads();

    // iterating over the subproblems defined by the coordinates'
    // ranges
    //
    // -|\
    // -|-\
    // -|--\
    // -|---\ i.e. A = (0,x), B = (y>0, z<size)
    // B|----\
    // -|-----\
    // -|------\
    // -+-------\
    // -> A

    for (register int a = 1; a < cities; a += OFFSET) {
        // if (a > cities - 2048)
        // a = cities - 2048;
        a = min (a, const1);
        rA_start = a - 1;
        rA_end = a + OFFSET;	// Vertical size
#ifndef USE_ORDERED_COORDS

        // copy the first range of coordinates
        for (register int k = rA_start + threadIdx.x; k < rA_end; k += blockDim.x) {
            coords_extended_A[k - rA_start] = coords_global[device_route[k]];
        }

#else

        for (register int k = rA_start + threadIdx.x; k < rA_end; k += blockDim.x) {
            coords_extended_A[k - rA_start] = coords_global[k];
        }

#endif

        for (register int b = const1; b > a - OFFSET; b -= OFFSET) {
            if ( (b < a) || (a == const1) ) {
                b = a;
            }

            rB_start = b - 1;
            rB_end = b + OFFSET;
#ifndef USE_ORDERED_COORDS

            // copy the second range of coordinates
            for (register int k = rB_start + threadIdx.x; k < rB_end;
                    k += blockDim.x) {
                coords_extended_B[k - rB_start] = coords_global[device_route[k]];
            }

#else

            for (register int k = rB_start + threadIdx.x; k < rB_end;
                    k += blockDim.x) {
                coords_extended_B[k - rB_start] = coords_global[k];
            }

#endif
            __syncthreads();
#ifdef LEGACY_GPUS
            iters_inside_a_blockB = 4;
#else
#ifndef K20
            iters_inside_a_blockB = 16;
#else
            iters_inside_a_blockB = 8;
#endif
#endif

            // check if there are some subblocks that can be skipped
            if (b == a)
                if (a != const1) {
                    iters_inside_a_blockB = const2;
                }

            // #if __CUDA_ARCH__ >= 300
#ifndef LEGACY_GPUS
#ifdef K20
#pragma unroll 8
#else
#pragma unroll 16
#endif
#endif

            // #endif
            // for a 2048x2048 problem size, each thread has to work 256 times...
            // (16x16)
            // since there are 16K threads running and 4,194,304 exchanges to be
            // checked
            for (register int re_i = 0; re_i < iters_inside_a_blockB; re_i++) {
                // #if __CUDA_ARCH__ >= 300
#ifndef LEGACY_GPUS
#ifdef K20
#pragma unroll 8
#else
#pragma unroll 16
#endif
#endif
                // #elif __CUDA_ARCH__ >= 210
                // seems to work with Fermi
                // #pragma unroll 4
                // #endif
#ifndef LEGACY_GPUS
#ifdef K20

                for (register int re_j = 0; re_j < 8; re_j++) {
#else

                for (register int re_j = 0; re_j < 16; re_j++) {
#endif
#else

                for (register int re_j = 0; re_j < 4; re_j++) {
#endif
#ifdef K20
                    // 256x256 block, if different this has to be changed
                    i = (local_id >> 8) + (re_i << 8);
                    j = mod256 (local_id) + (re_j << 8);
#else
                    // 128x128 block, if different this has to be changed
                    i = (local_id >> 7) + (re_i << 7);
                    j = mod128 (local_id) + (re_j << 7);
#endif

                    // only triangular matrix
                    if (i + b > j + a + 1) {
                        // calculate the effect of (i,j) swap
                        change =
                            calculateDistance2D_extended (j + 1, i + 1,
                                                          coords_extended_A,
                                                          coords_extended_B) +
                            calculateDistance2D_extended (i, j, coords_extended_B,
                                                          coords_extended_A) -
                            calculateDistance2D_extended (i, i + 1, coords_extended_B,
                                                          coords_extended_B) -
                            calculateDistance2D_extended (j, j + 1, coords_extended_A,
                                                          coords_extended_A);

                        if (change < best.minchange) {
                            best.minchange = change;
                            best.i = i + rB_start + 1;
                            best.j = j + rA_start + 1;
                        }
                    }
                }
            }

            __syncthreads();
        }
    }

    // intra-block reduction...it doesn't look pretty, but works
    best_values[threadIdx.x] = best;
    __syncthreads();
#ifndef LEGACY_GPUS

    // 1024 threads
    if (threadIdx.x < 512)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 512].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 512].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 512].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 512].j;
        }

    __syncthreads();

    // 512 threads
    if (threadIdx.x < 256)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 256].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 256].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 256].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 256].j;
        }

    __syncthreads();
#endif

    // 256 threads
    if (threadIdx.x < 128)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 128].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 128].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 128].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 128].j;
        }

    __syncthreads();

    // 128 threads
    if (threadIdx.x < 64)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 64].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 64].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 64].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 64].j;
        }

    __syncthreads();

    // 64 threads
    if (threadIdx.x < 32)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 32].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 32].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 32].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 32].j;
        }

    __syncthreads();

    // 32 threads
    if (threadIdx.x < 16)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 16].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 16].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 16].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 16].j;
        }

    __syncthreads();

    // 16 threads
    if (threadIdx.x < 8)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 8].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 8].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 8].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 8].j;
        }

    __syncthreads();

    // 8 threads
    if (threadIdx.x < 4)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 4].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 4].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 4].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 4].j;
        }

    __syncthreads();

    // 4 threads
    if (threadIdx.x < 2)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 2].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 2].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 2].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 2].j;
        }

    __syncthreads();

    // 2 threads
    if (threadIdx.x < 1)
        if (best_values[threadIdx.x].minchange >
                best_values[threadIdx.x + 1].minchange) {
            best_values[threadIdx.x].minchange =
                best_values[threadIdx.x + 1].minchange;
            best_values[threadIdx.x].i = best_values[threadIdx.x + 1].i;
            best_values[threadIdx.x].j = best_values[threadIdx.x + 1].j;
        }

    __syncthreads();

    // 1 thread left
    // 1st thread in a block holds the best result
    if (threadIdx.x == 0) {
#ifndef REDUCE_ON_CPU

        // reduction on GPU
        if (best_values[threadIdx.x].minchange < best_2opt->minchange) {
            // inter-block reduction
            atomicMin (& (best_2opt->minchange), best_values[threadIdx.x].minchange);
            // synchronize
            lock (mutex);

            if (best_values[threadIdx.x].minchange == best_2opt->minchange) {
                best_2opt->i = best_values[threadIdx.x].i;
                best_2opt->j = best_values[threadIdx.x].j;
            }

            unlock (mutex);
        }

#else
        // reduction on host
        best_2opt[blockIdx.x] = best_values[threadIdx.x];
#endif
    }
}
